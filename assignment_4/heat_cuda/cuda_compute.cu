#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "compute.h" 

//TODO: use cuda restric ponter
//TODO: fix reduction print

// Just to be used by kernel tuner 
// #define BLOCK_DIM 32
// #define GRID_DIM_X 32
// #define GRID_DIM_Y 32 

int init = 0; 

__constant__ __device__ double c_cdir = 0.25 * M_SQRT2 / (M_SQRT2 + 1.0);
__constant__ __device__ double c_cdiag = 0.25 / (M_SQRT2 + 1.0);

#define FPOPS_PER_POINT_PER_ITERATION (                 \
        1     /* current point 1 mul */ +               \
        3 + 1 /* direct neighbors 3 adds + 1 mul */ +   \
        3 + 1 /* diagonal neighbors 3 adds + 1 mul */ + \
        2     /* final add */ +                         \
        1     /* difference old/new */                  \
)

static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        printf("cuda error [code %d] [%s]\n", result, hipGetErrorString(result));
        exit(1);
    }
}

static int index(int i, int j, int WIDTH) { return WIDTH*i + j; }

__device__ int _index(int row, int col, int WIDTH) { return (WIDTH)*(row+1) + (col+1); }

static void fill_report(size_t w, size_t h, double* dst, struct results* r, double global_maxdiff, int iter, struct timeval before, struct timeval after) {
    double tmin = INFINITY, tmax = -INFINITY;
    double sum = 0.0;
    int i, j;

    for (i = 1; i < h - 1 ; ++i) {
        for (j = 1; j < w - 1 ; ++j) {
            double v = dst[index(i,j,w)];
            sum += v;
            if (tmin > v) tmin = v;
            if (tmax < v) tmax = v;
        }
    }

    r->niter = iter;
    r->maxdiff = global_maxdiff;
    r->tmin = tmin;
    r->tmax = tmax;
    r->tavg = sum / ((w-2) * (h-2));
    r->time = (double)(after.tv_sec - before.tv_sec) + 
    (double)(after.tv_usec - before.tv_usec) / 1e6;
}

static void summary_matrix(size_t w, size_t h, const double *a) {
    int H;
    printf("################\n"); 
    H = 0;
    printf("%lf\t%lf\t%lf\t%lf  ....  %lf\t%lf\t%lf\t%lf\n",
        a[index(H,0,w)], a[index(H,1,w)], a[index(H,2,w)], a[index(H,3,w)],
        a[index(H, w-4,w)], a[index(H, w-3,w)], a[index(H, w-2,w)], a[index(H, w-1,w)]);
    H = 1;
    printf("%lf\t%lf\t%lf\t%lf  ....  %lf\t%lf\t%lf\t%lf\n",
        a[index(H,0,w)], a[index(H,1,w)], a[index(H,2,w)], a[index(H,3,w)],
        a[index(H, w-4,w)], a[index(H, w-3,w)], a[index(H, w-2,w)], a[index(H, w-1,w)]);
    H = 2;
    printf("%lf\t%lf\t%lf\t%lf  ....  %lf\t%lf\t%lf\t%lf\n",
        a[index(H,0,w)], a[index(H,1,w)], a[index(H,2,w)], a[index(H,3,w)],
        a[index(H, w-4,w)], a[index(H, w-3,w)], a[index(H, w-2,w)], a[index(H, w-1,w)]);
    H = 3;
    printf("%lf\t%lf\t%lf\t%lf  ....  %lf\t%lf\t%lf\t%lf\n",
        a[index(H,0,w)], a[index(H,1,w)], a[index(H,2,w)], a[index(H,3,w)],
        a[index(H, w-4,w)], a[index(H, w-3,w)], a[index(H, w-2,w)], a[index(H, w-1,w)]);

    printf("... \t ... \n");
    printf("... \t ... \n");
    
    H = h-4;
    printf("%lf\t%lf\t%lf\t%lf  ....  %lf\t%lf\t%lf\t%lf\n",
        a[index(H,0,w)], a[index(H,1,w)], a[index(H,2,w)], a[index(H,3,w)],
        a[index(H, w-4,w)], a[index(H, w-3,w)], a[index(H, w-2,w)], a[index(H, w-1,w)]);
    H = h-3;
    printf("%lf\t%lf\t%lf\t%lf  ....  %lf\t%lf\t%lf\t%lf\n",
        a[index(H,0,w)], a[index(H,1,w)], a[index(H,2,w)], a[index(H,3,w)],
        a[index(H, w-4,w)], a[index(H, w-3,w)], a[index(H, w-2,w)], a[index(H, w-1,w)]);
    H = h-2;
    printf("%lf\t%lf\t%lf\t%lf  ....  %lf\t%lf\t%lf\t%lf\n",
        a[index(H,0,w)], a[index(H,1,w)], a[index(H,2,w)], a[index(H,3,w)],
        a[index(H, w-4,w)], a[index(H, w-3,w)], a[index(H, w-2,w)], a[index(H, w-1,w)]);
    H = h-1;
    printf("%lf\t%lf\t%lf\t%lf  ....  %lf\t%lf\t%lf\t%lf\n",
        a[index(H,0,w)], a[index(H,1,w)], a[index(H,2,w)], a[index(H,3,w)],
        a[index(H, w-4,w)], a[index(H, w-3,w)], a[index(H, w-2,w)], a[index(H, w-1,w)]);
    printf("###################\n");
}

/**
 * Main cell update kernel. lauched with always 32 threads in both dimension for each block. The block
 * size is calculated based on input size. 
 *
*/
 __global__ void update_kernel(double* src, double* dst, const double* cond, size_t w, size_t h, const size_t maxiter, double* maxdiff) {
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y; 
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= h-2 || j >= w-2 ) { return; }

    /* Reduction of strength - pre computed indexes */
    const unsigned int prev_row_base = (i)*w; 
    const unsigned int row_base = prev_row_base+w;  
    const unsigned int next_row_base = row_base+w; 
    const unsigned int cell_base = row_base+j+1; 

    double weight = cond[cell_base];
    double restw = 1.0 - weight;
    double v, v_old;
    v_old = src[cell_base];

    v = weight * v_old +
    (
        src[next_row_base+j+1] + 
        src[prev_row_base+j+1] + 
        src[row_base+j+2] + 
        src[row_base+j]
        ) * (restw * c_cdir)
    +
    ( 
        src[prev_row_base+j] + 
        src[prev_row_base+j+2] +
        src[next_row_base+j] +
        src[next_row_base+j+2]
        ) * (restw * c_cdiag);

    dst[cell_base] = v;

    double diff = fabs(v - v_old);
    maxdiff[cell_base] = diff; 

    /* Mirror first and last column */
    if ( j == 0 ) {
        dst[row_base+w-1] = dst[row_base+1]; 
    }
    if ( j == w-3 ) {
        dst[row_base+0] = dst[row_base+w-2]; 
    }
}

/**
* DEPRECATED
* Kernel used to mirror the first and last column
* Deplyed in the following size: 
*     dim3 mirror_dim_grid(1, GRID_DIM_Y, 1); 
*     dim3 mirror_dim_block(2, BLOCK_DIM, 1); 
*/
__global__ void mirror_kernel(double* dst, size_t w, size_t h) {
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y; 
    unsigned int row_base = (i+1)*w; 
    if ( i >= h-2 ) { return; }

    /* swap firs and last column in parallel, if needed */
    if (threadIdx.x == 0) { 
        dst[row_base+0] = dst[row_base+w-2]; 
    }   
    if (threadIdx.x == 1) {
        dst[row_base+w-1] = dst[row_base+1]; 
    }
}


__global__ void maxdiff_kernel_shared(size_t w, size_t h, double* maxdiff) {
    const unsigned int i = blockIdx.y * blockDim.y + threadIdx.y; 
    const unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int block_resp_index = blockDim.x*threadIdx.y + threadIdx.x; 

    if ( i >= h-2 || j >= w-2 ) { return; }

    /* Load the entire matrix in parallel and sync each thread */
    extern __shared__ double shared_maxdiff[]; 
    shared_maxdiff[block_resp_index] = maxdiff[_index(i,j,w)];     
    __syncthreads(); 

    /* Reduce each row of the block horizantally */
    for (unsigned int s=(blockDim.x)/2; s>0; s>>=1) {
        if (threadIdx.x < s) {
            if (shared_maxdiff[block_resp_index+s] > shared_maxdiff[block_resp_index]) {
                shared_maxdiff[block_resp_index] = shared_maxdiff[block_resp_index+s]; 
            }
        }
        __syncthreads();
    }

    /* Reduce the first column vertically */
    if ( threadIdx.x == 0 ) {
        for (unsigned int s=(blockDim.y)/2; s>0; s>>=1) {
            if (threadIdx.y < s) {
                if (shared_maxdiff[blockDim.x*(threadIdx.y+s)] > shared_maxdiff[block_resp_index]) {
                    shared_maxdiff[block_resp_index] = shared_maxdiff[blockDim.x*(threadIdx.y+s)]; 
                }
            }
            __syncthreads();
        }
    }

    /* one thread writes the result back */
    if ( threadIdx.x == 0 && threadIdx.y == 0 ) {
        maxdiff[w+1+(blockIdx.x)+(gridDim.x*blockIdx.y)] = shared_maxdiff[0]; 
    }
}

__global__ void maxdiff_kernel_shared_2(size_t w, size_t h, double* maxdiff) {
    // const unsigned int i = blockIdx.y * blockDim.y + threadIdx.y; 
    const unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int block_resp_index = blockDim.x*threadIdx.y + threadIdx.x; 

    /* Load the entire matrix in parallel and sync each thread */
    extern __shared__ double shared_maxdiff[]; 
    shared_maxdiff[threadIdx.x] = maxdiff[_index(0,j,w)];     
    __syncthreads(); 

    /* Reduce one row of the block horizantally */
    for (unsigned int s=(blockDim.x)/2; s>0; s>>=1) {
        if (threadIdx.x < s) {
            if (shared_maxdiff[block_resp_index+s] > shared_maxdiff[block_resp_index]) {
                shared_maxdiff[block_resp_index] = shared_maxdiff[block_resp_index+s]; 
            }
        }
        __syncthreads();
    }

    /* one thread writes the result back */
    if ( threadIdx.x == 0 && threadIdx.y == 0 ) {
        maxdiff[w+1] = shared_maxdiff[0]; 
    }
}

__global__ void maxdiff_kernel(size_t w, size_t h, double* maxdiff) {
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y; 
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= h-2 || j >= w-2 ) { return; }

    for (unsigned int s=(blockDim.x*gridDim.x)/2; s>0; s>>=1) {
        // if ( threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0 )
            // printf("Thread %d %d will compare %d to %d \n", i, j, j, j+s);

        if (j < s) {
            if (maxdiff[_index(i,j+s,w)] > maxdiff[_index(i,j,w)]) {
                maxdiff[_index(i,j,w)] = maxdiff[_index(i,j+s,w)]; 
            }
        }
        __syncthreads();
    }
}

__global__ void maxdiff_kernel_2(size_t w, size_t h, double* maxdiff) {
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y; 
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= h-2 || j >= w-2 ) { return; }

    for (unsigned int s=(blockDim.y*gridDim.y)/2; s>0; s>>=1) {
        if (i < s) {
            // if ( threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0 )
            //     printf("2Thread 0 0 0 will compare %d to %d \n", i, i+s);
            if (maxdiff[_index(i+s,j,w)] > maxdiff[_index(i,j,w)]) {
                maxdiff[_index(i,j,w)] = maxdiff[_index(i+s,j,w)]; 
            }
        }
        __syncthreads();
    }
}

extern "C" void cuda_do_compute(const struct parameters* p, struct results *r) {
    struct timeval before, after;

    const size_t N = p->N; 
    const size_t M =  p->M; 

    printf("ORIGINAL DIMENSTIONS [%zd %zd]\n", N, M); 

    const size_t w = M+2; 
    const size_t h = N+2; 

    const size_t MALLOC_VAL = w*h; 

    /* It is important for thread batches to be multiplies of 32 */
    unsigned const int BLOCK_DIM = 32; 
    unsigned int GRID_DIM_X; 
    unsigned int GRID_DIM_Y;

    /* Find the minimum number of blocks that is bigger than the data size */
    GRID_DIM_X = ceil((float)M/BLOCK_DIM); 
    GRID_DIM_Y = ceil((float)N/BLOCK_DIM);  

    const double *tinit = p->tinit; 
    const double *cond = p->conductivity;

    double *h_dst   = (double*)malloc(MALLOC_VAL*sizeof(double)); 
    double *h_src   = (double*)malloc(MALLOC_VAL*sizeof(double)); 
    double *h_cond  = (double*)malloc(MALLOC_VAL*sizeof(double)); 

    if (!h_src || !h_dst || !h_cond) {
        printf("malloc failed\n");
        exit(1);
    }

    double *d_src;
    double *d_dst;  
    double *d_cond; 

    /* Initialize value, mirrors, halo grids and stuff */
    int i, j;
    for (i = 1; i < h - 1; ++i) {
        for (j = 1; j < w - 1; ++j)
        {
            h_src[index(i,j,w)]  = tinit[index(i-1,j-1,M)];
            h_dst[index(i,j,w)]  = tinit[index(i-1,j-1,M)];
            h_cond[index(i,j,w)] = cond [index(i-1,j-1,M)];
        }
    }

    /* smear outermost row to border */
    for (j = 1; j < w-1; ++j) {
        h_src[0 + j] = h_src[0 + j] = h_src[1*w + j];
        h_dst[0 + j] = h_dst[0 + j] = h_dst[1*w + j];
        h_src[(h-1)*w + j] = h_src[(h-1)*w + j] = h_src[(h-2)*w + j];
        h_dst[(h-1)*w + j] = h_dst[(h-1)*w + j] = h_dst[(h-2)*w + j];
    }

    // mirror 
    for (i = 0; i < h ; ++i) {
        // column w-1 
        h_src[i*w + w-1] = h_src[i*w + 1];
        h_dst[i*w + w-1] = h_dst[i*w + 1];
        // column 0 
        h_src[i*w + 0]   = h_src[i*w + w-2];
        h_dst[i*w + 0]   = h_dst[i*w + w-2];
    }
    
    printf("GRID_DIM= [%d %d] | BLOCK_DIM=[%d %d] | SAHRED_MEM_SIZE %ld\n", GRID_DIM_X, GRID_DIM_Y, BLOCK_DIM, BLOCK_DIM, BLOCK_DIM*BLOCK_DIM*sizeof(double));

    dim3 update_dim_grid(GRID_DIM_X, GRID_DIM_Y, 1); 
    dim3 update_dim_block(BLOCK_DIM, BLOCK_DIM, 1); 

    dim3 mirror_dim_grid(1, GRID_DIM_Y, 1); 
    dim3 mirror_dim_block(2, BLOCK_DIM, 1); 

    //TODO BLOCK_DIM/2 is an optimization that only works with inputs of power of 2
    dim3 maxdiff_dim_grid(GRID_DIM_X, GRID_DIM_Y, 1); 
    // dim3 maxdiff_dim_block(BLOCK_DIM/2, BLOCK_DIM, 1); 
    dim3 maxdiff_dim_block(BLOCK_DIM, BLOCK_DIM, 1); 

    dim3 maxdiff_2_dim_grid(1, GRID_DIM_Y, 1); 
    dim3 maxdiff_2_dim_block(1, BLOCK_DIM, 1);

    dim3 maxdiff_2_shared_dim_grid(1, 1, 1); 
    dim3 maxdiff_2_shared_dim_block(GRID_DIM_X*GRID_DIM_Y, 1, 1); 

    /* start time */
    gettimeofday(&before, NULL); 

    /* Init space for src, dst, cond in GPU memeory */
    checkCudaCall(hipMalloc((void **) &d_src,  MALLOC_VAL*sizeof(double))); 
    checkCudaCall(hipMalloc((void **) &d_dst,  MALLOC_VAL*sizeof(double))); 
    checkCudaCall(hipMalloc((void **) &d_cond, MALLOC_VAL* sizeof(double))); 

    /* Copy everything to device memory */
    checkCudaCall(hipMemcpy(d_src, h_src,   MALLOC_VAL*sizeof(double), hipMemcpyHostToDevice)); 
    checkCudaCall(hipMemcpy(d_dst, h_dst,   MALLOC_VAL*sizeof(double), hipMemcpyHostToDevice)); 
    checkCudaCall(hipMemcpy(d_cond, h_cond, MALLOC_VAL*sizeof(double), hipMemcpyHostToDevice)); 

    /* maxdiff variables */
    double *h_maxdiff = (double *)malloc(MALLOC_VAL*sizeof(double)); 
    for (int i = 0; i < h*w; i++) { h_maxdiff[i] = 0; }
    
    double *d_maxdiff; 
    checkCudaCall(hipMalloc((void **) &d_maxdiff, MALLOC_VAL*sizeof(double))); 
    checkCudaCall(hipMemcpy(d_maxdiff, h_maxdiff, MALLOC_VAL*sizeof(double), hipMemcpyHostToDevice)); 

    double *global_maxdiff = (double*) malloc(sizeof(double)); 
    int it; 
    for (it = 0; it < p->maxiter; it++) {
        /* All cells will be updated in d_dest */
        update_kernel<<<update_dim_grid, update_dim_block>>>(d_src, d_dst, d_cond,w, h, p->maxiter, d_maxdiff); 

        /* update first and last column */
        mirror_kernel<<<mirror_dim_grid, mirror_dim_block>>>(d_dst, w, h); 

        /* calculate diff,  */
        // TODO: this is not correct for big sizes like 1000x1000, 2000x2000, instead works with 1024x1024 etc.
        // REASON: https://stackoverflow.com/questions/40402053/why-does-cuda-8-0-sometimes-have-a-bad-memory-access-while-7-5-doesnt
        // maxdiff_kernel<<<maxdiff_dim_grid, maxdiff_dim_block>>>(w, h, d_maxdiff);  
        // maxdiff_kernel_2<<<maxdiff_2_dim_grid, maxdiff_2_dim_block>>>(w, h, d_maxdiff);

        maxdiff_kernel_shared<<<maxdiff_dim_grid, maxdiff_dim_block, BLOCK_DIM*BLOCK_DIM*sizeof(double)>>>(w, h, d_maxdiff);
        maxdiff_kernel_shared_2<<<maxdiff_2_shared_dim_grid, maxdiff_2_shared_dim_block, GRID_DIM_X*GRID_DIM_Y*sizeof(double)>>>(w, h, d_maxdiff);

        /* Copy just one value from maxdiff kernel out */
        checkCudaCall(hipMemcpy(global_maxdiff, d_maxdiff+w+1, sizeof(double), hipMemcpyDeviceToHost)); 
        if ( *global_maxdiff < p->threshold ) { break; }

        if ( p->printreports ) { 
            if ( it % p->period == 0 ) {
                checkCudaCall(hipMemcpy(h_dst, d_dst, MALLOC_VAL*sizeof(double), hipMemcpyDeviceToHost));
                fill_report(w, h, h_dst, r, *global_maxdiff, it, before, after);  
                
                if (!init) {
                    printf("Output:\n\n"
                     "%13s %13s %13s %13s %13s %13s %13s\n",
                     "Iterations",
                     "T(min)", "T(max)", "T(diff)", "T(avg)", "Time", "FLOP/s");
                    init = 1;
                }
                gettimeofday(&after, NULL);  

                printf("%-13zu % .6e % .6e % .6e % .6e % .6e % .6e\n",
                 r->niter,
                 r->tmin,
                 r->tmax,
                 r->maxdiff,
                 r->tavg,
                 r->time,
                 (double)p->N * (double)p->M * 
                 (double)(r->niter * FPOPS_PER_POINT_PER_ITERATION +
                    (double)r->niter / p->period) / r->time);

            }
        }
        /* swap pointers for next iteration, if exists */
        // TODO: this will cause miskates for the last iteration in case of maxiter termination
        { double *tmp = d_src; d_src = d_dst; d_dst = tmp; }
    }


    /*  Sync and fetch the latest results */
    hipDeviceSynchronize();   
    checkCudaCall(hipGetLastError()); 
    checkCudaCall(hipMemcpy(h_dst, d_dst, MALLOC_VAL*sizeof(double), hipMemcpyDeviceToHost));
    gettimeofday(&after, NULL);  

    fill_report(w, h, h_dst, r, *global_maxdiff, it, before, after);     

    /* cleanup device */
    checkCudaCall(hipFree(d_dst)); 
    checkCudaCall(hipFree(d_cond)); 
    checkCudaCall(hipFree(d_src)); 
    checkCudaCall(hipFree(d_maxdiff)); 

    /* cleanup host */
    free(h_src);
    free(h_dst); 
    free(h_cond); 
    free(h_maxdiff);
}
