#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "compute.h" 

static const double c_cdir = 0.25 * M_SQRT2 / (M_SQRT2 + 1.0);
static const double c_cdiag = 0.25 / (M_SQRT2 + 1.0);

static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        printf("cuda error [code %d][%s]\n", result, hipGetErrorString(result));
        exit(1);
    }
}

static int index(int i, int j, int WIDTH) { return WIDTH*i + j; }

__device__ int _index(int row, int col, int WIDTH) { return (WIDTH)*(row+1) + (col+1); }

static void fill_report(size_t w, size_t h, double* dst, struct results* r, double global_maxdiff, int iter, struct timeval before, struct timeval after) {
    double tmin = INFINITY, tmax = -INFINITY;
    double sum = 0.0;
    int i, j;

    for (i = 1; i < h - 1 ; ++i) {
        for (j = 1; j < w - 1 ; ++j) {
            double v = dst[index(i,j,w)];
            sum += v;
            if (tmin > v) tmin = v;
            if (tmax < v) tmax = v;
        }
    }

    r->niter = iter;
    r->maxdiff = global_maxdiff;
    r->tmin = tmin;
    r->tmax = tmax;
    r->tavg = sum / ((w-2) * (h-2));
    r->time = (double)(after.tv_sec - before.tv_sec) + 
        (double)(after.tv_usec - before.tv_usec) / 1e6;
}

static void summary_matrix(size_t w, size_t h, const double *a) {
    int H;
    printf("################\n"); 
    H = 0;
    printf("%lf\t%lf\t%lf\t%lf  ....  %lf\t%lf\t%lf\t%lf\n",
        a[index(H,0,w)], a[index(H,1,w)], a[index(H,2,w)], a[index(H,3,w)],
        a[index(H, w-4,w)], a[index(H, w-3,w)], a[index(H, w-2,w)], a[index(H, w-1,w)]);
    H = 1;
    printf("%lf\t%lf\t%lf\t%lf  ....  %lf\t%lf\t%lf\t%lf\n",
        a[index(H,0,w)], a[index(H,1,w)], a[index(H,2,w)], a[index(H,3,w)],
        a[index(H, w-4,w)], a[index(H, w-3,w)], a[index(H, w-2,w)], a[index(H, w-1,w)]);
    H = 2;
    printf("%lf\t%lf\t%lf\t%lf  ....  %lf\t%lf\t%lf\t%lf\n",
        a[index(H,0,w)], a[index(H,1,w)], a[index(H,2,w)], a[index(H,3,w)],
        a[index(H, w-4,w)], a[index(H, w-3,w)], a[index(H, w-2,w)], a[index(H, w-1,w)]);
    H = 3;
    printf("%lf\t%lf\t%lf\t%lf  ....  %lf\t%lf\t%lf\t%lf\n",
        a[index(H,0,w)], a[index(H,1,w)], a[index(H,2,w)], a[index(H,3,w)],
        a[index(H, w-4,w)], a[index(H, w-3,w)], a[index(H, w-2,w)], a[index(H, w-1,w)]);

    printf("... \t ... \n");
    printf("... \t ... \n");
    
    H = h-4;
    printf("%lf\t%lf\t%lf\t%lf  ....  %lf\t%lf\t%lf\t%lf\n",
        a[index(H,0,w)], a[index(H,1,w)], a[index(H,2,w)], a[index(H,3,w)],
        a[index(H, w-4,w)], a[index(H, w-3,w)], a[index(H, w-2,w)], a[index(H, w-1,w)]);
    H = h-3;
    printf("%lf\t%lf\t%lf\t%lf  ....  %lf\t%lf\t%lf\t%lf\n",
        a[index(H,0,w)], a[index(H,1,w)], a[index(H,2,w)], a[index(H,3,w)],
        a[index(H, w-4,w)], a[index(H, w-3,w)], a[index(H, w-2,w)], a[index(H, w-1,w)]);
    H = h-2;
    printf("%lf\t%lf\t%lf\t%lf  ....  %lf\t%lf\t%lf\t%lf\n",
        a[index(H,0,w)], a[index(H,1,w)], a[index(H,2,w)], a[index(H,3,w)],
        a[index(H, w-4,w)], a[index(H, w-3,w)], a[index(H, w-2,w)], a[index(H, w-1,w)]);
    H = h-1;
    printf("%lf\t%lf\t%lf\t%lf  ....  %lf\t%lf\t%lf\t%lf\n",
        a[index(H,0,w)], a[index(H,1,w)], a[index(H,2,w)], a[index(H,3,w)],
        a[index(H, w-4,w)], a[index(H, w-3,w)], a[index(H, w-2,w)], a[index(H, w-1,w)]);
    printf("###################\n");
}
    
/**
 * Main cell update kernel. lauched with always 32 threads in both dimension for each block. The block
 * size is calculated based on input size. 
 *
*/
__global__ void cellUpdateKernel(double* src, double* dst, const double* cond, size_t w, size_t h, const size_t maxiter, double* maxdiff) {
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y; 
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= h-2 || j >= w-2 ) { return; }

    /* Reduction of strength - pre computed indexes */
    const unsigned int prev_row_base = (i)*w; 
    const unsigned int row_base = prev_row_base+w;  
    const unsigned int next_row_base = row_base+w; 
    const unsigned int cell_base = row_base+j+1; 

    double weight = cond[cell_base];
    double restw = 1.0 - weight;
    double v, v_old;
    v_old = src[cell_base];

    v = weight * v_old +
    (
        src[next_row_base+j+1] + 
        src[prev_row_base+j+1] + 
        src[row_base+j+2] + 
        src[row_base+j]
        ) * (restw * c_cdir)
    +
    ( 
        src[prev_row_base+j] + 
        src[prev_row_base+j+2] +
        src[next_row_base+j] +
        src[next_row_base+j+2]
        ) * (restw * c_cdiag);

    dst[cell_base] = v;

    double diff = fabs(v - v_old);
    maxdiff[cell_base] = diff; 
}

/**
* Kernel used to mirror the first and last column
* Deplyed in the following size: 
*     dim3 mirror_dim_grid(1, GRID_DIM_Y, 1); 
*     dim3 mirror_dim_block(2, BLOCK_DIM, 1); 
*/
__global__ void mirrorKernel(double* dst, size_t w, size_t h) {
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y; 
    unsigned int row_base = (i+1)*w; 
    if ( i >= h-2 ) { return; }

    /* swap firs and last column in parallel, if needed */
    if (threadIdx.x == 0) { 
        dst[row_base+0] = dst[row_base+w-2]; 
    }   
    if (threadIdx.x == 1) {
        dst[row_base+w-1] = dst[row_base+1]; 
    }
}


__global__ void diffUpdateKernel_sharedMem(size_t w, size_t h, double* maxdiff) {
    const unsigned int i = blockIdx.y * blockDim.y + threadIdx.y; 
    const unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int block_resp_index = blockDim.x*threadIdx.y + threadIdx.x; 

    if ( i >= h-2 || j >= w-2 ) { return; }

    /* Load the entire matrix in parallel and sync each thread */
    extern __shared__ double shared_maxdiff[]; 
    shared_maxdiff[block_resp_index] = maxdiff[_index(i,j,w)];     
    __syncthreads(); 

    /* Reduce each row of the block horizantally */
    for (unsigned int s=(blockDim.x)/2; s>0; s>>=1) {
        if (threadIdx.x < s) {
            if (shared_maxdiff[block_resp_index+s] > shared_maxdiff[block_resp_index]) {
                shared_maxdiff[block_resp_index] = shared_maxdiff[block_resp_index+s]; 
            }
        }
        __syncthreads();
    }

    /* Reduce the first column vertically */
    if ( threadIdx.x == 0 ) {
        for (unsigned int s=(blockDim.y)/2; s>0; s>>=1) {
            if (threadIdx.y < s) {
                if (shared_maxdiff[blockDim.x*(threadIdx.y+s)] > shared_maxdiff[block_resp_index]) {
                    shared_maxdiff[block_resp_index] = shared_maxdiff[blockDim.x*(threadIdx.y+s)]; 
                }
            }
            __syncthreads();
        }
    }

    /* one thread writes the result back */
    if ( threadIdx.x == 0 && threadIdx.y == 0 ) {
        maxdiff[w+1+(blockIdx.x)+(gridDim.x*blockIdx.y)] = shared_maxdiff[0]; 
    }
}

__global__ void diffUpdateKernel_sharedMem_2(size_t w, size_t h, double* maxdiff) {
    // const unsigned int i = blockIdx.y * blockDim.y + threadIdx.y; 
    const unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int block_resp_index = blockDim.x*threadIdx.y + threadIdx.x; 

    /* Load the entire matrix in parallel and sync each thread */
    extern __shared__ double shared_maxdiff[]; 
    shared_maxdiff[threadIdx.x] = maxdiff[_index(0,j,w)];     
    __syncthreads(); 

    /* Reduce one row of the block horizantally */
    for (unsigned int s=(blockDim.x)/2; s>0; s>>=1) {
        if (threadIdx.x < s) {
            if (shared_maxdiff[block_resp_index+s] > shared_maxdiff[block_resp_index]) {
                shared_maxdiff[block_resp_index] = shared_maxdiff[block_resp_index+s]; 
            }
        }
        __syncthreads();
    }

    /* one thread writes the result back */
    if ( threadIdx.x == 0 && threadIdx.y == 0 ) {
        maxdiff[w+1] = shared_maxdiff[0]; 
    }
}

__global__ void diffUpdateKernel(size_t w, size_t h, double* maxdiff) {
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y; 
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= h-2 || j >= w-2 ) { return; }

    for (unsigned int s=(blockDim.x*gridDim.x)/2; s>0; s>>=1) {
        // if ( threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0 )
        // printf("Thread 0 0 0 will compare %d to %d \n", j, j+s);
        if (j < s) {
            if (maxdiff[_index(i,j+s,w)] > maxdiff[_index(i,j,w)]) {
                maxdiff[_index(i,j,w)] = maxdiff[_index(i,j+s,w)]; 
            }
        }
        __syncthreads();
    }
}

__global__ void diffUpdateKernel_2(size_t w, size_t h, double* maxdiff) {
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y; 
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= h-2 || j >= w-2 ) { return; }

    for (unsigned int s=(blockDim.y*gridDim.y)/2; s>0; s>>=1) {
        if (i < s) {
        //     if ( threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0 )
        // printf("2Thread 0 0 0 will compare %d to %d \n", i, i+s);
            if (maxdiff[_index(i+s,j,w)] > maxdiff[_index(i,j,w)]) {
                maxdiff[_index(i,j,w)] = maxdiff[_index(i+s,j,w)]; 
            }
        }
        __syncthreads();
    }
}

__global__ void GlobalcellUpdateKernel(double* src, double* dst, const double* cond, size_t w, size_t h, const size_t maxiter, int* block_flag) {
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y; 
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
    int it; 
    for (it = 1; it < maxiter; it++) {
        double weight = cond[_index(i,j,w)];
        double restw = 1.0 - weight;
        double v, v_old;
        v_old = src[_index(i,j,w)];

        v = weight * v_old +
        (
            src[_index(i+1,j,w)] + 
            src[_index(i-1,j,w)] + 
            src[_index(i,j+1,w)] + 
            src[_index(i,j-1,w)]
            ) * (restw * c_cdir)
        +
        ( 
            src[_index(i-1,j-1,w)] + 
            src[_index(i-1,j+1,w)] +
            src[_index(i+1,j-1,w)] +
            src[_index(i+1,j+1,w)]
            ) * (restw * c_cdiag);

        dst[_index(i,j,w)] = v; 

        /* swap firs and last column in parallel, if needed */
        if (blockIdx.x == 0 && threadIdx.x == 0) { 
            dst[_index(i, -1, w)] = dst[_index(i, w-3, w)]; 
        }   
        if ( threadIdx.x == (blockDim.x-1) && blockIdx.x == (gridDim.x-1) ) {
            dst[_index(i, w-2, w)] = dst[_index(i, 0, w)]; 
        }

        /* Ensure that all threads in the current block are done with the update */
        __syncthreads(); 

        /* Ensure all perations in this block are written to memory */
        __threadfence_block(); 

        /* mark the iteration flag if this block */
        if ( threadIdx.x == 0 && threadIdx.y == 0 ) {
            // printf("Master thread of block %d %d. DONE\n", blockIdx.y, blockIdx.x);
            // TODO: this sould not be atomic, right? 
            block_flag[blockIdx.x + blockIdx.y * gridDim.x] = it; 
        }

        /* wait until all blocks are merked */
        while(1) {
            for (int p = 0; p < gridDim.x*gridDim.y; p++) {
                if (block_flag[p] != it ) { 
                    continue; 
                }
            } 
            break; 
        }

        printf("Thread %d done with iteration\n", it);
        /* all threads swap the pointers for the next iteration */
        { double *tmp = src; src = dst; dst = tmp; }
        // break;

    }
}


extern "C" void cuda_do_compute(const struct parameters* p, struct results *r) {
    struct timeval before, after;

    const size_t N = p->N; 
    const size_t M =  p->M; 

    printf("ORIGINAL DIMENSTIONS [%zd %zd]\n", N, M); 

    const size_t w = M+2; 
    const size_t h = N+2; 

    const size_t MALLOC_VAL = w*h; 

    /* It is important for thread batches to be multiplies of 32 */
    unsigned const int BLOCK_DIM = 32; 
    unsigned int GRID_DIM_X; 
    unsigned int GRID_DIM_Y;

    /* Find the minimum number of blocks that is bigger than the data size */
    GRID_DIM_X = ceil((float)M/BLOCK_DIM); 
    GRID_DIM_Y = ceil((float)N/BLOCK_DIM);  

    const double *tinit = p->tinit; 
    const double *cond = p->conductivity;

    double *h_dst   = (double*)malloc(MALLOC_VAL*sizeof(double)); 
    double *h_src   = (double*)malloc(MALLOC_VAL*sizeof(double)); 
    double *h_cond  = (double*)malloc(MALLOC_VAL*sizeof(double)); 

    if (!h_src || !h_dst || !h_cond) {
        printf("malloc failed\n");
        exit(1);
    }

    double *d_src;
    double *d_dst;  
    double *d_cond; 

    /* Initialize value, mirrors, halo grids and stuff */
    int i, j;
    for (i = 1; i < h - 1; ++i) {
        for (j = 1; j < w - 1; ++j)
        {
            h_src[index(i,j,w)]  = tinit[index(i-1,j-1,M)];
            h_dst[index(i,j,w)]  = tinit[index(i-1,j-1,M)];
            h_cond[index(i,j,w)] = cond [index(i-1,j-1,M)];
        }
    }

    /* smear outermost row to border */
    for (j = 1; j < w-1; ++j) {
        h_src[0 + j] = h_src[0 + j] = h_src[1*w + j];
        h_dst[0 + j] = h_dst[0 + j] = h_dst[1*w + j];
        h_src[(h-1)*w + j] = h_src[(h-1)*w + j] = h_src[(h-2)*w + j];
        h_dst[(h-1)*w + j] = h_dst[(h-1)*w + j] = h_dst[(h-2)*w + j];
    }

    // mirror 
    for (i = 0; i < h ; ++i) {
        // column w-1 
        h_src[i*w + w-1] = h_src[i*w + 1];
        h_dst[i*w + w-1] = h_dst[i*w + 1];
        // column 0 
        h_src[i*w + 0]   = h_src[i*w + w-2];
        h_dst[i*w + 0]   = h_dst[i*w + w-2];
    }
    
    printf("GRID_DIM= [%d %d] | BLOCK_DIM=[%d %d] | SAHRED_MEM_SIZE %ld\n", GRID_DIM_X, GRID_DIM_Y, BLOCK_DIM, BLOCK_DIM, BLOCK_DIM*BLOCK_DIM*sizeof(double));

    dim3 update_dim_grid(GRID_DIM_X, GRID_DIM_Y, 1); 
    dim3 update_dim_block(BLOCK_DIM, BLOCK_DIM, 1); 

    dim3 mirror_dim_grid(1, GRID_DIM_Y, 1); 
    dim3 mirror_dim_block(2, BLOCK_DIM, 1); 

    dim3 maxdiff_dim_grid(GRID_DIM_X, GRID_DIM_Y, 1); 
    dim3 maxdiff_dim_block(BLOCK_DIM, BLOCK_DIM, 1); 

    dim3 maxdiff_2_dim_grid(1, GRID_DIM_Y, 1); 
    dim3 maxdiff_2_dim_block(1, BLOCK_DIM, 1);

    dim3 maxdiff_2_shared_dim_grid(1, 1, 1); 
    dim3 maxdiff_2_shared_dim_block(GRID_DIM_X*GRID_DIM_Y, 1, 1); 

    /* start time */
    gettimeofday(&before, NULL); 

    /* Init space for src, dst, cond in GPU memeory */
    checkCudaCall(hipMalloc((void **) &d_src,  MALLOC_VAL*sizeof(double))); 
    checkCudaCall(hipMalloc((void **) &d_dst,  MALLOC_VAL*sizeof(double))); 
    checkCudaCall(hipMalloc((void **) &d_cond, MALLOC_VAL* sizeof(double))); 

    /* Copy everything to device memory */
    checkCudaCall(hipMemcpy(d_src, h_src,   MALLOC_VAL*sizeof(double), hipMemcpyHostToDevice)); 
    checkCudaCall(hipMemcpy(d_dst, h_dst,   MALLOC_VAL*sizeof(double), hipMemcpyHostToDevice)); 
    checkCudaCall(hipMemcpy(d_cond, h_cond, MALLOC_VAL*sizeof(double), hipMemcpyHostToDevice)); 

    /* maxdiff variables */
    double *h_maxdiff = (double *)malloc(MALLOC_VAL*sizeof(double)); 
    for (int i = 0; i < h*w; i++) { h_maxdiff[i] = 0; }
    double *d_maxdiff; 
    checkCudaCall(hipMalloc((void **) &d_maxdiff, MALLOC_VAL*sizeof(double))); 
    checkCudaCall(hipMemcpy(d_maxdiff, h_maxdiff, MALLOC_VAL*sizeof(double), hipMemcpyHostToDevice)); 

    double *global_maxdiff = (double*) malloc(sizeof(double)); 
    int it; 
    for (it = 0; it < p->maxiter; it++) {
        /* All cells will be updated in d_dest */
        cellUpdateKernel<<<update_dim_grid, update_dim_block>>>(
            d_src, d_dst, d_cond,
            w, h, p->maxiter, d_maxdiff); 

        /* update first and last column */
        // TODO: should be faster with two kernels with no IF inside? 
        mirrorKernel<<<mirror_dim_grid, mirror_dim_block>>>(d_dst, w, h); 

        /* calculate diff,  */
        // TODO: maybe would be more optimzied with an initial kernel half of the size in each row (see slides) 

        // diffUpdateKernel
        //     <<<maxdiff_dim_grid, maxdiff_dim_block>>>
        //     (w, h, d_maxdiff);  
        // diffUpdateKernel_2
        //     <<<maxdiff_2_dim_grid, maxdiff_2_dim_block>>>
        //     (w, h, d_maxdiff);

        diffUpdateKernel_sharedMem
            <<<maxdiff_dim_grid, maxdiff_dim_block, BLOCK_DIM*BLOCK_DIM*sizeof(double)>>>
            (w, h, d_maxdiff);

        diffUpdateKernel_sharedMem_2
            <<<maxdiff_2_shared_dim_grid, maxdiff_2_shared_dim_block, GRID_DIM_X*GRID_DIM_Y*sizeof(double)>>>
            (w, h, d_maxdiff);

        // DEBUG 
        // printf("result at end of iter %d\n", it);
        // checkCudaCall(hipMemcpy(h_dst, d_dst, MALLOC_VAL*sizeof(double), hipMemcpyDeviceToHost)); 
        // summary_matrix(w, h, h_dst);
        // checkCudaCall(hipMemcpy(h_maxdiff, d_maxdiff, w*h*sizeof(double), hipMemcpyDeviceToHost)); 
        // printf("maxdiff at end of iter %d\n", it);
        // summary_matrix(w, h, h_maxdiff);

        /* Copy just one value from maxdiff kernel out */
        checkCudaCall(hipMemcpy(global_maxdiff, d_maxdiff+w+1, sizeof(double), hipMemcpyDeviceToHost)); 
        if ( *global_maxdiff < p->threshold ) { break; }

        if ( p->printreports ) { 
            if ( it % p->period == 0 ) {
                checkCudaCall(hipMemcpy(h_dst, d_dst, MALLOC_VAL*sizeof(double), hipMemcpyDeviceToHost));
                fill_report(w, h, h_dst, r, *global_maxdiff, it, before, after);  
                // report_results(p, r);
            }
        }
        /* swap pointers for next iteration, if exists */
        // TODO: this will cause miskates for the last iteration in case of maxiter termination
        { double *tmp = d_src; d_src = d_dst; d_dst = tmp; }
    }

    /*  Sync and fetch the latest results */
    hipDeviceSynchronize();   
    checkCudaCall(hipGetLastError()); 
    checkCudaCall(hipMemcpy(h_dst, d_dst, MALLOC_VAL*sizeof(double), hipMemcpyDeviceToHost));
    gettimeofday(&after, NULL);  
    // summary_matrix(w, h, h_dst);
    fill_report(w, h, h_dst, r, *global_maxdiff, it, before, after);     

    /* cleanup device */
    checkCudaCall(hipFree(d_dst)); 
    checkCudaCall(hipFree(d_cond)); 
    checkCudaCall(hipFree(d_src)); 
    checkCudaCall(hipFree(d_maxdiff)); 

    /* cleanup host */
    free(h_src);
    free(h_dst); 
    free(h_cond); 
    free(h_maxdiff);
}
