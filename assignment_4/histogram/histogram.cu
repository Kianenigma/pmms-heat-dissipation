#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <getopt.h>
#include <math.h>
#include <string.h>
#include <iostream>
#include "timer.h"

using namespace std;

#define hist_size 256

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}


__global__ void histogramKernel(unsigned char* image, long img_size, unsigned int* histos) {
    __shared__ unsigned int shared_histo[hist_size];
    unsigned int tid = threadIdx.x;
    unsigned int i = tid + blockDim.x * blockIdx.x;

    // initialize shared memory to 0 in parallel (256 first threads in each block)
    if(tid < hist_size) {
        shared_histo[tid] = 0;
    }
    // make sure, that all writes to shared memory are finished
    __syncthreads();

    if(i < img_size) {
        atomicAdd(&shared_histo[image[i]], 1);
    }
    // make sure, that all writes to shared memory are finished
    __syncthreads();

    // write histogram of block back to global memory
    if(tid < hist_size) {
        // advance pointer to histograms to block specific one
        histos += blockIdx.x * hist_size;
        histos[tid] = shared_histo[tid];
    }
}

void histogramCuda(unsigned char* image, long img_size, unsigned int* histogram) {
    int threadBlockSize = 512;
    int blocks;

    // calculate number of blocks based on img_size
    blocks = img_size / threadBlockSize;
    if(img_size % threadBlockSize != 0) {
        blocks++;
    }
    printf("Blocks: %d\n", blocks);

    // allocate the vectors on the GPU
    unsigned char* deviceImage = NULL;
    checkCudaCall(hipMalloc((void **) &deviceImage, img_size * sizeof(unsigned char)));
    if (deviceImage == NULL) {
        cout << "could not allocate memory!" << endl;
        return;
    }
    unsigned int* deviceHistos = NULL;
    checkCudaCall(hipMalloc((void **) &deviceHistos, blocks * hist_size * sizeof(unsigned int)));
    if (deviceHistos == NULL) {
        checkCudaCall(hipFree(deviceImage));
        cout << "could not allocate memory!" << endl;
        return;
    }

    timer kernelTime1 = timer("kernelTime1");
    timer memoryTime = timer("memoryTime");

    // copy the original vectors to the GPU
    memoryTime.start();
    checkCudaCall(hipMemcpy(deviceImage, image, img_size*sizeof(unsigned char), hipMemcpyHostToDevice));
    memoryTime.stop();

    // execute kernel
    kernelTime1.start();
    histogramKernel<<<blocks, threadBlockSize>>>(deviceImage, img_size, deviceHistos);
    hipDeviceSynchronize();
    kernelTime1.stop();

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // TODO: get rid of temp_histograms
    unsigned int* temp_histograms = (unsigned int *)malloc(blocks * hist_size * sizeof(unsigned int));  

    // copy result back
    memoryTime.start();
    checkCudaCall(hipMemcpy(temp_histograms, deviceHistos, blocks * hist_size * sizeof(unsigned int), hipMemcpyDeviceToHost));
    memoryTime.stop();

    // TODO: remove reduction
    for(int b=0; b<hist_size; b++) {
        histogram[b] = 0;
    }

    for(int i=0; i<blocks;i++) {
        for(int b=0; b<hist_size; b++) {
            histogram[b] += temp_histograms[(i*hist_size)+b];
        }
    }

    //printf("\n\n b0=%d\n\n", b0);

    checkCudaCall(hipFree(deviceImage));
    checkCudaCall(hipFree(deviceHistos));

    cout << "histogram (kernel): \t\t" << kernelTime1  << endl;
    cout << "histogram (memory): \t\t" << memoryTime << endl;
    cout << "histogram total: \t\t  = " << (kernelTime1.getTimeInSeconds() + memoryTime.getTimeInSeconds()) << " seconds" << endl;
}


__global__ void histogramKernelSimple(unsigned char* image, long img_size, unsigned int* histogram) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i < img_size) {
        atomicAdd(&histogram[image[i]], 1);
    }
}

void histogramCudaSimple(unsigned char* image, long img_size, unsigned int* histogram) {
    int threadBlockSize = 512;
    int blocks;

    // calculate number of blocks based on img_size
    blocks = img_size / threadBlockSize;
    if(img_size % threadBlockSize != 0) {
        blocks++;
    }
    printf("Blocks: %d\n", blocks);

    // allocate the vectors on the GPU
    unsigned char* deviceImage = NULL;
    checkCudaCall(hipMalloc((void **) &deviceImage, img_size * sizeof(unsigned char)));
    if (deviceImage == NULL) {
        cout << "could not allocate memory!" << endl;
        return;
    }
    unsigned int* deviceHisto = NULL;
    checkCudaCall(hipMalloc((void **) &deviceHisto, hist_size * sizeof(unsigned int)));
    if (deviceHisto == NULL) {
        checkCudaCall(hipFree(deviceImage));
        cout << "could not allocate memory!" << endl;
        return;
    }

    timer kernelTime1 = timer("kernelTime");
    timer memoryTime = timer("memoryTime");

    // copy the original vectors to the GPU
    memoryTime.start();
    checkCudaCall(hipMemcpy(deviceImage, image, img_size*sizeof(unsigned char), hipMemcpyHostToDevice));
    checkCudaCall(hipMemset(deviceHisto, 0, hist_size * sizeof(unsigned int)));
    memoryTime.stop();

    // execute kernel
    kernelTime1.start();
    histogramKernelSimple<<<blocks, threadBlockSize>>>(deviceImage, img_size, deviceHisto);
    hipDeviceSynchronize();
    kernelTime1.stop();

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // copy result back
    memoryTime.start();
    checkCudaCall(hipMemcpy(histogram, deviceHisto, hist_size * sizeof(unsigned int), hipMemcpyDeviceToHost));
    memoryTime.stop();

    checkCudaCall(hipFree(deviceImage));
    checkCudaCall(hipFree(deviceHisto));

    cout << "histogram simple (kernel): \t" << kernelTime1  << endl;
    cout << "histogram simple (memory): \t" << memoryTime << endl;
    cout << "histogram simple total: \t  = " << (kernelTime1.getTimeInSeconds() + memoryTime.getTimeInSeconds()) << " seconds" << endl;
}

void histogramSeq(unsigned char* image, long img_size, unsigned int* histogram) {
  int i; 

  timer sequentialTime = timer("Sequential");
  
  for (i=0; i<hist_size; i++) histogram[i]=0;

  sequentialTime.start();
  for (i=0; i<img_size; i++) {
	histogram[image[i]]++;
  }
  sequentialTime.stop();
  
  cout << "histogram (seq): \t\t" << sequentialTime << endl;
}


/*
    make clean && make && prun -v -1 -np 1 -native '-C GTX480 --gres=gpu:1' ./myhistogram
    -s executes simple histogram kernel, default=advanced kernel
    -l size of histgram image, default=655360
*/
int main(int argc, char* argv[]) {
    int c;
    long img_size = 655360;
    int simple = 0;


    while((c = getopt(argc, argv, "l:s")) != -1) {
        switch(c) {
            case 'l':
                img_size = atoi(optarg);
                break;
            case 's':
                simple = 1;
                break;
            case '?':
                if(optopt == 'l') {
                    fprintf(stderr, "Option -%c requires an argument.\n", optopt);
                }
                else if(isprint(optopt)) {
                    fprintf(stderr, "Unknown option '-%c'.\n", optopt);
                }
                else {
                    fprintf(stderr, "Unknown option character '\\x%x'.\n", optopt);
                }
                return -1;
            default:
                return -1;
        }
    }

    unsigned char *image = (unsigned char *)malloc(img_size * sizeof(unsigned char)); 
    unsigned int *histogramS = (unsigned int *)malloc(hist_size * sizeof(unsigned int));     
    unsigned int *histogram = (unsigned int *)malloc(hist_size * sizeof(unsigned int));

    // initialize the vectors.
    for(long i=0; i<img_size; i++) {
        image[i] = (unsigned char) (rand() % hist_size);
    }

    cout << "Compute the histogram of a gray image with " << img_size << " pixels." << endl;

    histogramSeq(image, img_size, histogramS);

    if(simple == 1) {
        histogramCudaSimple(image, img_size, histogram);
    } else {
        histogramCuda(image, img_size, histogram);  
    }

    // verify the results
    for(int i=0; i<hist_size; i++) {
	  if (histogram[i]!=histogramS[i]) {
            cout << "error in results! Bin " << i << " is "<< histogram[i] << ", but should be " << histogramS[i] << endl; 
            exit(1);
        }
    }
    cout << "results OK!" << endl;
     
    free(image);
    free(histogram);
    free(histogramS);         
    
    return 0;
}
